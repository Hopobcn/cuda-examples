#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <cassert>
#include <auto/auto.hpp>
#include <util/grid_stride.hpp>
#include <util/cuda_grid_config.hpp>
#include <util/cuda_error.hpp>
#include <util/cuda_init.hpp>
#include <cub/hipcub/hipcub.hpp>

using cuda::grid_stride_range;
using cuda::util::getGridDimensions;
using cuda::util::lang::range;


template <typename T>
__global__
void streaming(const T* x, const T* y, T* z, unsigned N, T alpha) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    printf("blockIdx.x %d - blockDim.x %d - threadIdx.x %d = %d\n",
           blockIdx.x, blockDim.x, threadIdx.x, i);
    /*
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {
        //auto lambda = [&](T x, T y, T alpha) { return alpha * x + y; };

        //z[i] = lambda(x[i],y[i],alpha)


    }*/
}

template <typename T>
void lambda_test(cuda::device& gpu, unsigned N) {
    try {
        cuda::device gpu;
        cuda::error err;

        cout << "--- testing lambda ---" << endl;

        size_t free, total, allocated_by_os;
        gpu.getMemInfo(free, total);
        allocated_by_os = total - free;

        T *px, *py, *pz;
        err = hipMalloc((void**)&px, N * sizeof(T));
        err = hipMalloc((void**)&py, N * sizeof(T));
        err = hipMalloc((void**)&pz, N * sizeof(T));
        T alpha = 0.8;

        err = hipMemset(px, 3, N * sizeof(T));
        err = hipMemset(py, 2, N * sizeof(T));

        gpu.getMemInfo(free, total);
        cout << "Free mem: " << free/(1024*1024) << " / " << total/(1024*1024) << " MB" << endl;

        size_t est_program_alloc = 3 * N * sizeof(T);
        size_t real_program_alloc = total - free - allocated_by_os;
        float  factor = (real_program_alloc - est_program_alloc)/ static_cast<float>(real_program_alloc);

        cout << "Mem allocated by os: "             << allocated_by_os/(1024*1024)      << " MB" << endl;
        cout << "Mem allocated by program[Real]: "  << real_program_alloc/(1024*1024)   << " MB" << endl;
        cout << "Mem allocated by program[Est]: "   << est_program_alloc/(1024*1024)    << " MB" << endl;
        cout << "Difference "                       << factor << endl;

        unsigned block_size_x = 128;
        unsigned block_size_y = 1;
        unsigned block_size_z = 1;
        dim3 dimGrid = getGridDimensions(N, 1, 1, block_size_x, block_size_y, block_size_z);
        dim3 dimBlock( block_size_x, block_size_y, block_size_z );

        std::cout << "Launching streaming test" << std::endl;
        std::cout << "Grid [" << dimGrid.x << "," << dimGrid.y << "," << dimGrid.z << "]" << std::endl;

        streaming<T><<<dimGrid, dimBlock>>>(px, py, pz, N, alpha);

        err = hipFree(px);
        err = hipFree(py);
        err = hipFree(pz);

    } catch(cuda::cuda_exception error) {
        std::cout << error.what() << std::endl;
    }
}

void cuda_cpp11_testing(cuda::device& gpu, unsigned N)
{
    lambda_test<float>(gpu, N);

}