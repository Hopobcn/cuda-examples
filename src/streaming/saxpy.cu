#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cassert>
#include <cstdio>
#include <streaming/saxpy.hpp>
#include <util/grid_stride.hpp>
#include <util/cuda_grid_config.hpp>
#include <util/cuda_error.hpp>

using cuda::grid_stride_range;
using cuda::util::lang::range;


template <typename T>
__global__
void saxpy_gpu_c_array(const T* x, const T* y, T* z, unsigned N, T alpha) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
             i < N;
             i += blockDim.x * gridDim.x) {
        z[i] = alpha * x[i] + y[i];
    }
}

template <typename T>
__global__
void saxpy_gpu_cpp_array(const T* x, const T* y, T* z, unsigned N, T alpha) {
    for (auto i : grid_stride_range<unsigned>(0, N) ) {
        z[i] = alpha * x[i] + y[i];
    }
}

template <typename T>
__global__
void saxpy_gpu_c_vector(const cuda::vector<T>& x, const cuda::vector<T>& y, cuda::vector<T>& z, unsigned N, T alpha) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {
        z[i] = alpha * x[i] + y[i];
    }
}

template <typename T>
__global__
void saxpy_gpu_cpp_vector(const cuda::vector<T>& x, const cuda::vector<T>& y, cuda::vector<T>& z, unsigned N, T alpha) {
    for (auto i : grid_stride_range<unsigned>(0, N) ) {
        z[i] = alpha * x[i] + y[i];
    }
}


template <typename T, const int unroll>
__global__
void saxpy_gpu_c_vector_unroll(const T* x, const T* y, T* z, unsigned N, T alpha) {
    T x_reg[unroll], y_reg[unroll];
    unsigned i;
    for ( i = unroll * blockIdx.x * blockDim.x + threadIdx.x;
          i < N - unroll * blockDim.x * gridDim.x;
          i +=    unroll * blockDim.x * gridDim.x ) {
        #pragma unroll
        for (int j = 0; j < unroll; j++) {
            unsigned gindex = j * blockDim.x + i;
            x_reg[j] = x[gindex];
            y_reg[j] = y[gindex];
        }
        #pragma unroll
        for (int j = 0; j < unroll; j++) {
            unsigned gindex = j * blockDim.x + i;
            z[gindex] = alpha * x_reg[j] + y_reg[j];
        }
    }
    // to avoid the (index<N) conditional in the inner loop,
    // we left off some work at the end
    for (int j = 0; j < unroll; j++) {
        #pragma unroll
        for (int j = 0; j < unroll; j++) {
            unsigned gindex = j * blockDim.x + i;
            if (gindex < N) {
                x_reg[j] = x[gindex];
                y_reg[j] = y[gindex];
            }
        }
        #pragma unroll
        for (int j = 0; j < unroll; j++) {
            unsigned gindex = j * blockDim.x + i;
            if (gindex < N)
                z[gindex] = alpha * x_reg[j] + y_reg[j];
        }
    }
}

template <typename T, const int unroll>
__global__
void saxpy_gpu_cpp_vector_unroll(const cuda::vector<T>& x, const cuda::vector<T>& y, cuda::vector<T>& z, unsigned N, T alpha) {
    T x_reg[unroll], y_reg[unroll];
    auto i_last = 0;
    for (auto i : grid_stride_range<unsigned>(0, N, unroll) ) {
        for (auto j : range<unsigned>(0, unroll)) {
            unsigned gindex = j * blockDim.x + i;
            x_reg[j] = x[gindex];
            y_reg[j] = y[gindex];
        }
        for (auto j : range(0, unroll)) {
            unsigned gindex = j * blockDim.x + i;
            z[gindex] = alpha * x_reg[j] + y_reg[j];
        }
        i_last = i;
    }
    // to avoid the (index<N) conditional in the inner loop,
    // we left off some work at the end
    for (auto j : range<unsigned>(0, unroll)) {
        for (auto j : range<unsigned>(0, unroll)) {
            unsigned gindex = j * blockDim.x + i_last;
            if (gindex < N) {
                x_reg[j] = x[gindex];
                y_reg[j] = y[gindex];
            }
        }
        for (auto j : range<unsigned>(0, unroll) ) {
            unsigned gindex = j * blockDim.x + i_last;
            if (gindex < N)
                z[gindex] = alpha * x_reg[j] + y_reg[j];
        }
    }
}

void run_saxpy(const cuda::vector<float>& x,
               const cuda::vector<float>& y,
                     cuda::vector<float>& z,
               const float* px,
               const float* py,
                     float* pz,
               unsigned N,
               float alpha,
               unsigned repetitions) {
    using T = float;

    cuda::error err;
    unsigned block_size_x = 64;
    unsigned block_size_y = 1;
    unsigned block_size_z = 1;
    dim3 dimGrid = cuda::util::getGridDimensions(N, 1, 1, block_size_x, block_size_y, block_size_z);
    dim3 dimBlock( block_size_x, block_size_y, block_size_z );

    unsigned block_size2_x = 64;
    unsigned block_size2_y = 1;
    unsigned block_size2_z = 1;
    const unsigned unroll2 = 2;
    dim3 dimGrid2  = cuda::util::getGridDimensions(N/unroll2, 1, 1, block_size2_x, block_size2_y, block_size2_z);
    dim3 dimBlock2( block_size2_x, block_size2_y, block_size2_z );

    unsigned block_size4_x = 64;
    unsigned block_size4_y = 1;
    unsigned block_size4_z = 1;
    const unsigned unroll4 = 4;
    dim3 dimGrid4 = cuda::util::getGridDimensions(N/unroll4, 1, 1, block_size4_x, block_size4_y, block_size4_z);
    dim3 dimBlock4( block_size4_x, block_size4_y, block_size4_z );

    std::cout << "Launching saxpy kernels" << std::endl;
    std::cout << "Grid 1 [" << dimGrid.x << "," << dimGrid.y << "," << dimGrid.z << "]" << std::endl;
    std::cout << "Grid 2 [" << dimGrid2.x << "," << dimGrid2.y << "," << dimGrid2.z << "]" << std::endl;
    std::cout << "Grid 4 [" << dimGrid4.x << "," << dimGrid4.y << "," << dimGrid4.z << "]" << std::endl;

    std::cout << "Block 1 [" << dimBlock.x << "," << dimBlock.y << "," << dimBlock.z << "]" << std::endl;
    std::cout << "Block 2 [" << dimBlock2.x << "," << dimBlock2.y << "," << dimBlock2.z << "]" << std::endl;
    std::cout << "Block 4 [" << dimBlock4.x << "," << dimBlock4.y << "," << dimBlock4.z << "]" << std::endl;

    for (int i = 0; i < repetitions; i++) {
        saxpy_gpu_c_vector<T><<<dimGrid.x, dimBlock.x>>>(x, y, z, N, alpha);
        err = hipGetLastError();

        saxpy_gpu_cpp_vector<T><<<dimGrid.x, dimBlock>>>(x, y, z, N, alpha);
        err = hipGetLastError();

        saxpy_gpu_cpp_array<T><<<dimGrid.x, dimBlock>>>(px, py, pz, N, alpha);
        err = hipGetLastError();

        saxpy_gpu_c_array<T><<<dimGrid.x, dimBlock>>>(px, py, pz, N, alpha);
        err = hipGetLastError();


        saxpy_gpu_cpp_vector_unroll<T, unroll2><<<dimGrid2.x, dimBlock2>>>(x, y, z, N, alpha);
        err = hipGetLastError();
        saxpy_gpu_cpp_vector_unroll<T, unroll4><<<dimGrid4.x, dimBlock4>>>(x, y, z, N, alpha);
        err = hipGetLastError();


        saxpy_gpu_c_vector_unroll<T, unroll2><<<dimGrid2.x, dimBlock2>>>(px, py, pz, N, alpha);
        err = hipGetLastError();
        saxpy_gpu_c_vector_unroll<T, unroll4><<<dimGrid4.x, dimBlock4>>>(px, py, pz, N, alpha);
        err = hipGetLastError();
    }

    err = hipDeviceSynchronize();
}
